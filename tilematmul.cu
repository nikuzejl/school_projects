/** tilematmul.c
 *
 * Lab 11 Part 2, CPSC 375.
 * This program does basic matrix multiplication with CUDA.
 * This program uses the more sophisticated block matrix mult algorithm.
 *
 * Run as:
 * 	./tilematmul n
 *
 * where n is the size of the nxn matrix and is divisible by the tilesize, 16
 *
 * 		
 	    n	  Time
 	----------------------
        16	 0.024576 ms
       256	 1.771360 ms
       512	 14.059648 ms
      1024	 108.453156 ms
      2048	 686.185242 ms
      4096	 5158.414062 ms
      
 *
 * Author: Bettina, Lewis
 * Date: 12/9/2019
 *
 */
 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int TILE_WIDTH = 16;

__global__ void TiledMatrixMulKernel(double* M, double* N, double* P, int Width) {
	
	__shared__ double ds_M[TILE_WIDTH][TILE_WIDTH];
	__shared__ double ds_N[TILE_WIDTH][TILE_WIDTH];
	
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	double Pvalue = 0;
	
	// Loop over the M and N tiles required to compute the P element
	for (int ph = 0; ph < Width/TILE_WIDTH; ++ph) {
	
		// Collaborative loading of M and N tiles into shared memory
		ds_M[ty][tx] = M[Row*Width + ph*TILE_WIDTH + tx];
		ds_N[ty][tx] = N[(ph*TILE_WIDTH + ty)*Width + Col];
		__syncthreads();
	
		for (int i = 0; i < TILE_WIDTH; ++i)
			Pvalue += ds_M[ty][i] * ds_N[i][tx];
		
		__syncthreads();
	}
	
	P[Row*Width+Col] = Pvalue;
}

int main(int argc, char* argv[]) {

	int 				n = atoi(argv[1]);
	int 				size;
	double 				*a, *b, *c, *d_a, *d_b, *d_c;
	double				*ap, *bp;
	int				x;
	float 				elapsedTime;
        hipEvent_t 		        start, stop;
	
	// allocate space
	size = n*n*sizeof(double);
	a = (double *) malloc(size);
	b = (double *) malloc(size);
	c = (double *) malloc(size);
	
	// fill dummy matrices
	ap = a; 
	bp = b;
	for (x = 0; x < n*n; x++) {
		*ap++ = 1;
		*bp++ = 1;
	}
	
	
	// allocate device space
	hipMalloc((void**) &d_a, size);
	hipMalloc((void**) &d_b, size);
	hipMalloc((void**) &d_c, size);
	
	// copy input arrays to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	// start timer
	hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    
        // do matrix multiplication on device
        dim3 DimGrid((n-1)/16+1, (n-1)/16+1, 1);
	dim3 DimBlock(16,16,1);
	TiledMatrixMulKernel<<<DimGrid, DimBlock>>>(d_a, d_b, d_c, n); 
	
	// stop timer
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
	
	// copy out results
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
        // print results
        printf("%10d\t %f ms\n", n, elapsedTime);
	

	// free vars
	hipEventDestroy(start);
        hipEventDestroy(stop);
	free(a); free(b); free(c);
	
	
	return 0;
}
