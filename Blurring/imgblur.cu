#include "hip/hip_runtime.h"
/*
J. Lewis Nikuze
CSPC 375
Assignment 9

Blurring a picture using QDMBP and and CUDA
*/


 #include "qdbmp.h"
 #include <stdio.h>

 #define BLUR_SIZE   7
 #define CHANNELS    3	

__global__ void blurKernel(unsigned char * in, unsigned char * out, int w, int h) {
    int Col  = blockIdx.x * blockDim.x + threadIdx.x;
    int Row  = blockIdx.y * blockDim.y + threadIdx.y;
    if (Col < w && Row < h) {
        int pixVal = 0;
        int pixels = 0;
        // Get the average of the surrounding 2xBLUR_SIZE x 2xBLUR_SIZE box
        for (int blurRow = -BLUR_SIZE; blurRow< BLUR_SIZE+1; ++blurRow) {
            for (int blurCol = -BLUR_SIZE; blurCol< BLUR_SIZE+1; ++blurCol) {
                int curRow = Row + blurRow;
                int curCol = Col + blurCol; // Verify we have a valid image pixel
                if (curRow> -1 && curRow< h && curCol> -1 && curCol< w) {
                    pixVal += in[curRow* w + curCol];
                    pixels++; // Keep track of number of pixels in the accumulated total
                }
            }
        } // Write our new pixel value out
        out[Row * w + Col] = (unsigned char) (pixVal/ pixels);
    } 
}

int main( int argc, char* argv[] )
{       UCHAR r,g,b;
	UINT	width, height;
	BMP*	bmp;
        unsigned char *dev_in;
        unsigned char *dev_out;
	int x, y, size;
	unsigned char *array;

	/* Read an image file */
	bmp = BMP_ReadFile( argv[ 1 ] );
	BMP_CHECK_ERROR( stdout, -1 );

	/* Get image's dimensions */
	width = BMP_GetWidth( bmp );
	height = BMP_GetHeight( bmp );

        size = sizeof(unsigned char) *width*height*CHANNELS;

        hipMalloc((void**) &dev_in, size);           // allocates out on device 
        hipMalloc((void**) &dev_out,size);           // allocates in on device

	array = (unsigned char *) malloc(size);

        //mapping a bmp file to an array of unsigned chars
	for(x=0; x < width; x++){
   	  for(y= 0 ; y< height; y++){
       	    UCHAR r,g,b;
      	    BMP_GetPixelRGB(bmp,x,y,&r, &g, &b);

            array[x*CHANNELS*width + CHANNELS*y] = r;
            array[x*CHANNELS*width + CHANNELS*y+1] = g;
            array[x*CHANNELS*width + CHANNELS*y+2] = b;
         }
        }

        hipMemcpy(dev_in, array, size, hipMemcpyHostToDevice);

        dim3 DimGrid(((CHANNELS*width)-1)/16 + 1, (height-1)/16 + 1, 1);
        dim3 DimBlock(16, 16, 1);

        blurKernel<<<DimGrid, DimBlock>>>(dev_in, dev_out, width*3, height);

        hipMemcpy(array, dev_out, size, hipMemcpyDeviceToHost);

        //mapping the array to a bmp file
	for(x=0; x < width; x++){
  	  for(y= 0; y< height;y++){
     	    r = array[x*CHANNELS*width + CHANNELS*y ];
            g = array[x*CHANNELS*width + CHANNELS*y+1];
       	    b = array[x*CHANNELS*width + CHANNELS*y+2];

            BMP_SetPixelRGB(bmp,x,y,r,g,b);
          }
        }

	/* Save result */
	BMP_WriteFile( bmp, argv[ 2 ] );
	BMP_CHECK_ERROR( stdout, -2 );

	/* Free all memory allocated for the image */
	BMP_Free( bmp );

        hipFree(dev_in);
        hipFree(dev_out);

	return 0;
}
